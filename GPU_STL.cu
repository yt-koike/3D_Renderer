#include "hip/hip_runtime.h"
#include<stdio.h>
#include <string.h>
#include <unistd.h>
#include "headers/RenderSuite.h"
#include "headers/PPM.h"
#include "headers/STL.h"
#define POLY_INTERSECTION_GPU_H
int main(int argn,char** argv)
{
    Ray camera(Vec3(0, 0,-5), Vec3(0, 0,1));
    Scene scene(camera, Color(255, 255, 255), Color(100, 149, 237));
    // load cube from file
    Material coneMt(Color(Vec3(0.1)), Color(Vec3(0.69,0,0)), Color(Vec3(0.3)), Vec3(8));
    Polygon3D cone = STLBinLoad("STL/Cone.stl").move(Vec3(-1,0.1,5));
    cone.setMaterial(coneMt);
    scene.add(&cone);
    Polygon3D ICO = STLBinLoad("STL/ICO_Sphere.stl").move(Vec3(1,0.1,3));
    scene.add(&ICO);
    Polygon3D ICO2 = *ICO.copy();
    ICO2.move(Vec3(0,0,10));
    scene.add(&ICO2);
    Material mirrorMt(Color(Vec3(0.01)), Color(Vec3(0.1)), Color(Vec3(0.1)), Vec3(8));
    mirrorMt.setUsePerfectReflectance(1);
    mirrorMt.setCatadioptricFactor(Color(Vec3(0.7)));
    Sphere* sp = new Sphere(Vec3(0,1,10),0.5,mirrorMt);
    //scene.add(sp);
    printf("Load Complete.\n");

    scene.add(new Plane(Vec3(0,-1,0),Vec3(0,1,0)));

    // add lights
    scene.addLight(new PointLightSource(Vec3(0, 0, -5), Color(Vec3(1))));
    scene.addLight(new PointLightSource(Vec3(0, 20, 0), Color(Vec3(1))));

    int width,height;
    switch(argn){
        case 2:
        sscanf(argv[1],"%d",&width);
        sscanf(argv[1],"%d",&height);
        break;
        case 3:
        sscanf(argv[1],"%d",&width);
        sscanf(argv[2],"%d",&height);
        break;
        default:
        width = height = 256;
        break;
    }
    char filename[100];
    PPM ppmwriter(width, height, 255);
    clock_t st,ed;
    printf("Render Start. (%d x %d)\n",width, height);
    st = clock();
    ColorImage img = scene.draw(width, height);
    ed = clock();
    printf("Render End.\n");
    sprintf(filename, "GPU_STL.ppm");
    ppmwriter.import(img);
    ppmwriter.writePPM(filename);
    //irfanview(filename);
    return 0;
}
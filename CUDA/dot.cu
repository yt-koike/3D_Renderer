#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "device_atomic_functions.h"
#include <stdio.h>
#include <stdlib.h>
#define N (2048 * 8)
#define THREADS_PER_BLOCK 512

__global__ void dot(int *a, int *b, int *c)
{
    __shared__ int temp[THREADS_PER_BLOCK];
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    c[index] = a[index] * b[index];
}

int main()
{
    int *a, *b, *c;
    int *dev_a, *dev_b, *dev_c;
    int size = N * sizeof(int);

   //allocate space for the variables on the device
    hipMalloc((void **)&dev_a, size);
    hipMalloc((void **)&dev_b, size);
    hipMalloc((void **)&dev_c, size);

   //allocate space for the variables on the host
   a = (int *)malloc(size);
   b = (int *)malloc(size);
   c = (int *)malloc(size);

   //this is our ground truth
   int sumTest = 0;
   //generate numbers
   for (int i = 0; i < N; i++)
   {
       a[i] = rand() % 10;
       b[i] = rand() % 10;
       printf("%d * %d = %d \n",a[i],b[i],a[i]*b[i]);
   }

   hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
   hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

   dot<<< N / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> >(dev_a, dev_b, dev_c);

   hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);
   for (int i = 0; i < N; i++)
   {
       printf("%d * %d = %d \n",a[i],b[i],c[i]);
   }

   free(a);
   free(b);
   free(c);

   hipFree(a);
   hipFree(b);
   hipFree(c);

   //system("pause");

   return 0;

 }